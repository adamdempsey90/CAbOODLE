#include "caboodle.h"
#include <hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
extern "C" void dev2host(Field *fld) {
    gpuErrchk( hipMemcpy( fld->cpu, fld->gpu, sizeof(double)*(fld->size),hipMemcpyDeviceToHost)) 
}
extern "C" void host2dev(Field *fld) {
    gpuErrchk( hipMemcpy( fld->gpu, fld->cpu, sizeof(double)*(fld->size),hipMemcpyHostToDevice)) 
}

extern "C" void all2host(void) {
    dev2host(dens);
    dev2host(pres);
    dev2host(vx);
    dev2host(vy);
    dev2host(vz);
}
extern "C" void all2dev(void) {
    host2dev(dens);
    host2dev(pres);
    host2dev(vx);
    host2dev(vy);
    host2dev(vz);
}

extern "C" void reset_field(Field *fld) {
    int i;
    for(i=0;i<fld->size;i++) fld->cpu[i] = 0;
    host2dev(fld);
}

extern "C" Field *init_field(int size_x, int size_y, int size_z) {
    Field *fld;
    double *arr_cpu;
    double *arr_gpu;
    fld = (Field *)malloc(sizeof(Field));

    int size = size_x*size_y*size_z;

    arr_cpu = (double *)malloc(sizeof(double)*size);
    if (arr_cpu == NULL) {
        printf("Not enough space on CPU\n");
        exit(1);
    }
    /* Should use hipMallocPitch instead */
    gpuErrchk(hipMalloc((void **) &arr_gpu, size_x*size_y*size_z*sizeof(double)))
    if (arr_gpu == NULL) {
        printf("Not enough space on GPU\n");
        exit(1);
    }

    fld->size_x = size_x;
    fld->size_y = size_y;
    fld->size_z = size_z;
    fld->size = size;
    fld->cpu = arr_cpu;
    fld->gpu = arr_gpu;
    reset_field(fld);
    return fld;
}
extern "C" void free_field(Field *fld) {
    free(fld->cpu);
    gpuErrchk(hipFree(fld->gpu));
    free(fld);
}
